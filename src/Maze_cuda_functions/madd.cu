#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_episodes) {
        float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
        exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
    }
}

void epsilonGreedyCUDA(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    float* d_exploration_rates;
    
    hipStream_t stream = 0;

    hipStreamCreate(&stream);

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice, stream);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    // Launch kernel with specified stream
    epsilonGreedyKernel << <grid, block, 0, stream >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost, stream);

    hipFree(d_exploration_rates);


    hipStreamSynchronize(stream);  // Synchronize with the stream

    hipStreamDestroy(stream);
}

__global__ void randomArrayKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * width + idx_x;

    __shared__ int shared_start_x, shared_start_y, shared_end_x, shared_end_y;

    // Only one thread in the block initializes shared variables
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_start_x = start_x;
        shared_start_y = start_y;
        shared_end_x = end_x;
        shared_end_y = end_y;
    }

    // Synchronize to make sure shared variables are initialized before use
    __syncthreads();

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Use shared_start_x, shared_start_y, shared_end_x, shared_end_y instead of start_x, start_y, end_x, end_y
    maze_array[idx] = hiprand_uniform(&state) < 0.3 ? 0 : 1;
    // Avoid global memory access in the loop
    //if (idx_x == shared_start_x && idx_y == shared_start_y)
    maze_array[shared_start_y * width + shared_start_x] = 2;

    //if (idx_x == shared_end_x && idx_y == shared_end_y)
    maze_array[shared_end_y * width + shared_end_x] = 3;
}

void randomArrayCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void randomizeZerosKernel(int* A, int size, float percentage, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * size + idx_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    if (A[idx] == 1 && hiprand_uniform(&state) < percentage) {
        A[idx] = 0;
    }
}

void randomizeZerosCuda(int* A, int X, int Y, float percentage, unsigned long long seed) {
    int* d_A;

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((X + block.x - 1) / block.x, (Y + block.y - 1) / block.y);

    int size = X * Y * sizeof(int);

    hipMalloc((void**)&d_A, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    randomizeZerosKernel << <grid, block >> > (d_A, X, percentage, seed);

    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipDeviceSynchronize();
}

__device__ void custom_swap(int& a, int& b) {
    int temp = a;
    a = b;
    b = temp;
}

__global__ void dfs_kernel(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int current_x = blockIdx.x * blockDim.x + threadIdx.x;
    int current_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = current_y * width + current_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Check if the current cell is the end cell
    if (current_x == end_x && current_y == end_y) {
        // You've reached the end, you can handle it as needed
        return;
    }

    // Define the possible moves (right, left, down, up)
    int moves[4][2] = { {0, 1}, {0, -1}, {1, 0}, {-1, 0} };

    // Fisher-Yates shuffle to traverse randomly
    for (int i = 4; i > 0; --i) {
        int j = hiprand_uniform(&state) * (i + 1);

        // Custom swap moves[i] with moves[j]
        custom_swap(moves[i][0], moves[j][0]);
        custom_swap(moves[i][1], moves[j][1]);
    }

    // Check each possible move
    int stack[50000];  // Use an array as a stack (adjust the size as needed)
    int stack_top = -1; // Stack top initialization

    for (int i = 0; i < 4; ++i) {
        int new_x = current_x + moves[i][0];
        int new_y = current_y + moves[i][1];

        // Check if the new position is within bounds
        if (new_x >= 0 && new_x < width && new_y >= 0 && new_y < height) {
            int new_idx = new_y * width + new_x;

            // Check if the new cell is open and not visited
            if (maze_array[new_idx] == 1) {
                if (hiprand_uniform(&state) < 0.45) {
                    maze_array[new_idx] = 0;
                }

                // Push onto the stack
                stack[++stack_top] = new_idx;

                // If the end has been reached, exit the loop
                if (new_x == end_x && new_y == end_y) {
                    return;
                }
            }
        }
    }

    // Pop from the stack and iterate
    while (stack_top >= 0) {
        int popped_idx = stack[stack_top--];
        int popped_x = popped_idx % width;
        int popped_y = popped_idx / width;

        // Continue DFS from the popped position
        dfs_kernel << <1, 1 >> > (maze_array, width, height, popped_x, popped_y, end_x, end_y, seed);

        // If the end has been reached in the recursive call, exit the loop
        if (maze_array[end_y * width + end_x] == 4) {
            return;
        }
    }
}

void dfsCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;
    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    dfs_kernel << <grid, block >> > (d_maze_array, width, height, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void guaranteePathKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int start_idx = start_y * width + start_x;
    int end_idx = end_y * width + end_x;

    hiprandState_t state;
    hiprand_init(seed, idx_x + idx_y * width, 0, &state);

     //Set cells in the same row or column as start or end to 0 for only half of the width or height
    if (idx_y == start_y || idx_y == end_y) {
        int start_col = (idx_y == start_y) ? 0 : width / 4;
        int end_col = (idx_y == end_y) ? width : width / 4;
        for (int i = start_col; i < end_col; ++i) {
            maze_array[idx_y * width + i] = 0;
        }
    }

    if (idx_x == start_x || idx_x == end_x) {
        int start_row = (idx_x == start_x) ? 0 : height / 4;
        int end_row = (idx_x == end_x) ? height : height / 4;
        for (int i = start_row; i < end_row; ++i) {
            maze_array[i * width + idx_x] = 0;
        }
    }

    // Randomly select two additional spots and apply the same logic using hiprand
    if (hiprand_uniform(&state) < 0.003) {
        int rand_x1 = hiprand(&state) % width;
        int rand_y1 = hiprand(&state) % height;
        for (int i = 0; i < width / 2; ++i) {
            maze_array[rand_y1 * width + i] = 0;
        }

        int rand_x2 = hiprand(&state) % width;
        int rand_y2 = hiprand(&state) % height;
        for (int i = width / 2; i < width; ++i) {
            maze_array[rand_y2 * width + i] = 0;
        }
    }

    maze_array[start_idx] = 3;
    maze_array[end_idx] = 4;
}

void guranteePathCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    guaranteePathKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void copyKernel(int* maze_array, int* shared_array, int shared_width, int shared_height, int width, int height, unsigned long long seed) {
    __shared__ int shared_data[3][3];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int size = width * height;

    // Initialize shared_data with 1's in the outside and 0's in the middle and one random spot
    if (threadIdx.x < 3 && threadIdx.y < 3)
    {
        if (threadIdx.x == 1 && threadIdx.y == 1)
        {
            shared_data[threadIdx.y][threadIdx.x] = 2; // Middle value is 0
        }
        else
        {
            shared_data[threadIdx.y][threadIdx.x] = 1; // Outside values are 1
        }
    }

    __syncthreads(); // Synchronize threads to make sure shared_data is populated

    hiprandState_t state;
    hiprand_init(seed + tid, tid, 0, &state);

    // Iterate through maze_array and copy shared_data to random indexes based on the random_value
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
    {
        // Generate a random value
        float random_value = hiprand_uniform(&state);

        // Copy shared_data to maze_array based on the random_value
        if (random_value < 0.01)
        {
            int start_index_x = i % (width - shared_width + 1);
            int start_index_y = (i / width) % (height - shared_height + 1);

            for (int j = 0; j < shared_height; ++j)
            {
                for (int k = 0; k < shared_width; ++k)
                {
                    maze_array[(start_index_y + j) * width + (start_index_x + k)] = shared_data[j][k];
                }
            }
        }
    }
}

void copyCuda(int* maze_array, int* shared_array, int shared_width, int shared_height, int width, int height, unsigned long long seed) {
    // Declare device arrays
    int* d_maze_array;
    int* d_shared_array;

    int maze_size = width * height;
    int shared_size = shared_width * shared_height;

    // Allocate device memory
    hipMalloc((void**)&d_maze_array, sizeof(int) * maze_size);
    hipMalloc((void**)&d_shared_array, sizeof(int) * shared_size);

    // Copy data from host to device
    hipMemcpy(d_maze_array, maze_array, sizeof(int) * maze_size, hipMemcpyHostToDevice);
    hipMemcpy(d_shared_array, shared_array, sizeof(int) * shared_size, hipMemcpyHostToDevice);

    // Set up grid and block sizes
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Launch the kernel
    copyKernel << < grid, block >> > (d_maze_array, d_shared_array, shared_width, shared_height, width, height, seed);

    // Copy the result back to the host
    hipMemcpy(maze_array, d_maze_array, sizeof(int) * maze_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_maze_array);
    hipFree(d_shared_array);
    hipDeviceSynchronize();
}

////////////////////////Control functions

//epsilonGreedykernel_non_async
__global__ void epsilonGreedyKernel_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {

int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < num_episodes) {
    float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
    exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
}
}

void epsilonGreedyCUDA_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    float* d_exploration_rates;

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));
    hipMemcpy(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    epsilonGreedyKernel << <grid, block >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    hipMemcpy(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_exploration_rates);
    hipDeviceSynchronize();
}

//randomArrayKernel_non_shared
__global__ void randomArrayKernel_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

int idx = idx_y * width + idx_x;

hiprandState state;
hiprand_init(seed, idx, 0, &state);

// Set the maze value randomly
maze_array[idx] = hiprand_uniform(&state) < 0.4 ? 0 : 1;
maze_array[start_y * width + start_x] = 2;
maze_array[end_y * width + end_x] = 3;

}

void randomArrayCuda_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}


