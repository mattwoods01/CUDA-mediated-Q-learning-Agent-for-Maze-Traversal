#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end);
__global__ void randomArrayKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);
__global__ void randomizeZerosKernel(int* array, int size, float percentage, unsigned long long seed);
__global__ void dfs_kernel(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);
__global__ void guaranteePathKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);

__global__ void randomArrayKernel_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);
__global__ void epsilonGreedyKernel_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end);
//__global__ void dfs_kernel_ctrl(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);

__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_episodes) {
        float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
        exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
    }
}

void epsilonGreedyCUDA(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end, hipStream_t stream = 0) {
    float* d_exploration_rates;

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice, stream);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    // Launch kernel with specified stream
    epsilonGreedyKernel << <grid, block, 0, stream >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost, stream);

    hipFree(d_exploration_rates);
}

__global__ void randomArrayKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * width + idx_x;

    __shared__ int shared_start_x, shared_start_y, shared_end_x, shared_end_y;

    // Only one thread in the block initializes shared variables
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_start_x = start_x;
        shared_start_y = start_y;
        shared_end_x = end_x;
        shared_end_y = end_y;
    }

    // Synchronize to make sure shared variables are initialized before use
    __syncthreads();

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Use shared_start_x, shared_start_y, shared_end_x, shared_end_y instead of start_x, start_y, end_x, end_y
    maze_array[idx] = hiprand_uniform(&state) < 0.3 ? 0 : 1;
    // Avoid global memory access in the loop
    //if (idx_x == shared_start_x && idx_y == shared_start_y)
    maze_array[shared_start_y * width + shared_start_x] = 2;

    //if (idx_x == shared_end_x && idx_y == shared_end_y)
    maze_array[shared_end_y * width + shared_end_x] = 3;
}

void randomArrayCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void randomizeZerosKernel(int* A, int size, float percentage, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * size + idx_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    if (A[idx] == 1 && hiprand_uniform(&state) < percentage) {
        A[idx] = 5;
    }
}

void randomizeZerosCuda(int* A, int X, int Y, float percentage, unsigned long long seed) {
    int* d_A;

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((X + block.x - 1) / block.x, (Y + block.y - 1) / block.y);

    int size = X * Y * sizeof(int);

    hipMalloc((void**)&d_A, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    randomizeZerosKernel << <grid, block >> > (d_A, X, percentage, seed);

    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipDeviceSynchronize();
}

__device__ void swap(int& a, int& b) {
    int temp = a;
    a = b;
    b = temp;
}

__global__ void dfs_kernel(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int current_x = blockIdx.x * blockDim.x + threadIdx.x;
    int current_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = current_y * width + current_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Check if the current cell is the end cell
    if (current_x == end_x && current_y == end_y) {
        // You've reached the end, you can handle it as needed
        return;
    }

    // Define the possible moves (right, left, down, up)
    int moves[4][2] = { {0, height / 3}, {0, -height / 3}, {width / 3, 0}, {-width / 3, 0} };

    // Fisher-Yates shuffle to traverse randomly
    for (int i = 3; i > 0; --i) {
        int j = hiprand_uniform(&state) * (i + 1);

        // Swap moves[i] with moves[j]
        swap(moves[i][0], moves[j][0]);
        swap(moves[i][1], moves[j][1]);
    }

    // Check each possible move
    for (int i = 0; i < 4; ++i) {
        int new_x = current_x + moves[i][0];
        int new_y = current_y + moves[i][1];

        // Check if the new position is within bounds
        if (new_x >= 0 && new_x < width && new_y >= 0 && new_y < height) {
            int new_idx = new_y * width + new_x;


            // Check if the new cell is open and not visited
            if (maze_array[new_idx] == 1) {
                if (hiprand_uniform(&state) < 0.1) {
                    maze_array[new_idx] = 0;
                }
                // Recursively call DFS on the new cell
                dfs_kernel << < 1, 1 >> > (maze_array, width, height, start_x, start_y, end_x, end_y, seed);

                // If the end has been reached in the recursive call, exit the loop
                if (maze_array[end_y * width + end_x] == 4) {
                    return;
                }
            }
        }
    }

    // Additional condition to prevent changing the starting cell

}

__global__ void guaranteePathKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int start_idx = start_y * width + start_x;
    int end_idx = end_y * width + end_x;

    hiprandState_t state;
    hiprand_init(seed, idx_x + idx_y * width, 0, &state);

    // Set cells in the same row or column as start or end to 0 for only half of the width or height
    if (idx_y == start_y || idx_y == end_y) {
        int start_col = (idx_y == start_y) ? 0 : width / 2;
        int end_col = (idx_y == end_y) ? width : width / 2;
        for (int i = start_col; i < end_col; ++i) {
            maze_array[idx_y * width + i] = 0;
        }
    }

    if (idx_x == start_x || idx_x == end_x) {
        int start_row = (idx_x == start_x) ? 0 : height / 2;
        int end_row = (idx_x == end_x) ? height : height / 2;
        for (int i = start_row; i < end_row; ++i) {
            maze_array[i * width + idx_x] = 0;
        }
    }

    maze_array[start_idx] = 3;
    maze_array[end_idx] = 4;

    // Randomly select two additional spots and apply the same logic using hiprand
    if (hiprand_uniform(&state) < 0.005) {
        int rand_x1 = hiprand(&state) % width;
        int rand_y1 = hiprand(&state) % height;
        for (int i = 0; i < width / 2; ++i) {
            maze_array[rand_y1 * width + i] = 0;
        }

        int rand_x2 = hiprand(&state) % width;
        int rand_y2 = hiprand(&state) % height;
        for (int i = width / 2; i < width; ++i) {
            maze_array[rand_y2 * width + i] = 0;
        }
    }
}

void guranteePathCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    guaranteePathKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

///////////////////////////Control functions

//epsilonGreedykernel_non_async
__global__ void epsilonGreedyKernel_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {

int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < num_episodes) {
    float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
    exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
}
}

void epsilonGreedyCUDA_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    float* d_exploration_rates;

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));
    hipMemcpy(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    epsilonGreedyKernel << <grid, block >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    hipMemcpy(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_exploration_rates);
    hipDeviceSynchronize();
}

//randomArrayKernel_non_shared
__global__ void randomArrayKernel_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

int idx = idx_y * width + idx_x;

hiprandState state;
hiprand_init(seed, idx, 0, &state);

// Set the maze value randomly
maze_array[idx] = hiprand_uniform(&state) < 0.4 ? 0 : 1;
maze_array[start_y * width + start_x] = 2;
maze_array[end_y * width + end_x] = 3;

}

void randomArrayCuda_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}


void dfsCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;
    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    dfs_kernel << <grid, block >> > (d_maze_array, width, height, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}