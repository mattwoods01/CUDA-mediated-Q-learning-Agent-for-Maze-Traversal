#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    __shared__ int shared_num_episodes;  // Declare shared memory variable

    if (threadIdx.x == 0) {
        shared_num_episodes = num_episodes;  // Store num_episodes in shared memory
    }

    __syncthreads();  // Ensure all threads have stored num_episodes in shared memory

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < shared_num_episodes) {
        float frac = static_cast<float>(tid) / static_cast<float>(shared_num_episodes);
        exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
    }
}

void epsilonGreedyCUDA(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    float* d_exploration_rates;

    hipStream_t stream = 0;

    hipStreamCreate(&stream);

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice, stream);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    // Launch kernel with specified stream and shared memory size
    epsilonGreedyKernel << <grid, block, num_episodes * sizeof(int), stream >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost, stream);

    hipFree(d_exploration_rates);

    hipStreamSynchronize(stream);  // Synchronize with the stream

    hipStreamDestroy(stream);
}

__global__ void randomArrayKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * width + idx_x;

    __shared__ int shared_start_x, shared_start_y, shared_end_x, shared_end_y;

    // Only one thread in the block initializes shared variables
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_start_x = start_x;
        shared_start_y = start_y;
        shared_end_x = end_x;
        shared_end_y = end_y;
    }

    // Synchronize to make sure shared variables are initialized before use
    __syncthreads();

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Use shared_start_x, shared_start_y, shared_end_x, shared_end_y instead of start_x, start_y, end_x, end_y
    maze_array[idx] = hiprand_uniform(&state) < 0.45 ? 0 : 1;
    // Avoid global memory access in the loop
    //if (idx_x == shared_start_x && idx_y == shared_start_y)
    maze_array[shared_start_y * width + shared_start_x] = 2;

    //if (idx_x == shared_end_x && idx_y == shared_end_y)
    maze_array[shared_end_y * width + shared_end_x] = 3;
}

void randomArrayCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void randomizeZerosKernel(int* A, int size, float percentage, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = idx_y * size + idx_x;
    int s_idx = threadIdx.y * blockDim.x + threadIdx.x;

    // Define shared memory matrix
    extern __shared__ int s_matrix[];

    // Load data into shared memory
    s_matrix[s_idx] = A[idx];

    __syncthreads(); // Ensure all data is loaded

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Perform operations using shared memory
    if (s_matrix[s_idx] == 1 && hiprand_uniform(&state) < percentage) {
        s_matrix[s_idx] = 0;
    }

    __syncthreads(); // Ensure all operations are done

    // Write data back to global memory
    A[idx] = s_matrix[s_idx];
}

void randomizeZerosCuda(int* A, int X, int Y, float percentage, unsigned long long seed) {
    int* d_A;
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((X + block.x - 1) / block.x, (Y + block.y - 1) / block.y);
    int size = X * Y * sizeof(int);

    hipMalloc((void**)&d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    // Calculate shared memory size per block
    int sharedMemSize = dimx * dimy * sizeof(int);

    // Launch the kernel with shared memory
    randomizeZerosKernel << <grid, block, sharedMemSize >> > (d_A, X, percentage, seed);

    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipDeviceSynchronize();
}

__device__ void custom_swap(int& a, int& b) {
    int temp = a;
    a = b;
    b = temp;
}

__global__ void dfs_kernel(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int current_x = blockIdx.x * blockDim.x + threadIdx.x;
    int current_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize hiprand state
    hiprandState state;
    int idx = current_y * width + current_x;
    hiprand_init(seed, idx, 0, &state);

    // Stack for storing nodes to visit
    int stack[1024][2];
    int stack_top = 0;

    // Push the start position onto the stack
    stack[stack_top][0] = start_x;
    stack[stack_top][1] = start_y;
    stack_top++;

    while (stack_top > 0) {
        // Pop the top node from the stack
        stack_top--;
        int current_x = stack[stack_top][0];
        int current_y = stack[stack_top][1];

        // Mark the current cell as visited
        int current_idx = current_y * width + current_x;
        maze_array[current_idx] = 0; // Or another value to mark as visited

        // Check if the current cell is the end cell
        if (current_x == end_x && current_y == end_y) {
            // End reached
            return;
        }

        // Define the possible moves (right, left, down, up)
        int moves[4][2] = { {0, 1}, {0, -1}, {1, 0}, {-1, 0} };

        // Fisher-Yates shuffle to traverse randomly
        for (int i = 3; i > 0; --i) {
            int j = hiprand_uniform(&state) * (i + 1);
            int temp_x = moves[i][0], temp_y = moves[i][1];
            moves[i][0] = moves[j][0];
            moves[i][1] = moves[j][1];
            moves[j][0] = temp_x;
            moves[j][1] = temp_y;
        }

        // Check each possible move
        for (int i = 0; i < 4; ++i) {
            int new_x = current_x + moves[i][0];
            int new_y = current_y + moves[i][1];

            // Check if the new position is within bounds
            if (new_x >= 0 && new_x < width && new_y >= 0 && new_y < height) {
                int new_idx = new_y * width + new_x;

                // Check if the new cell is open and not visited
                if (maze_array[new_idx] == 0) {
                    if (hiprand_uniform(&state) < .001) {
                        maze_array[current_idx] = 1;
                    }
                    
                    if (stack_top < 1024) {
                        stack[stack_top][0] = new_x;
                        stack[stack_top][1] = new_y;
                        stack_top++;
                    }
                }
            }
        }
    }
}

void dfsCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;
    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    dfs_kernel << <grid, block >> > (d_maze_array, width, height, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void guaranteePathKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (idx_x >= width || idx_y >= height) {
        return; // Out of bounds, do nothing
    }

    int start_idx = start_y * width + start_x;
    int end_idx = end_y * width + end_x;

    hiprandState_t state;
    hiprand_init(seed, idx_x + idx_y * width, 0, &state);

    // Set cells in the same row as start or end to 0
    if (idx_y == start_y || idx_y == end_y) {
        for (int i = 0; i < width; ++i) {
            maze_array[idx_y * width + i] = 0;
        }
    }

    // Set cells in the same column as start or end to 0
    if (idx_x == start_x || idx_x == end_x) {
        for (int i = 0; i < height; ++i) {
            maze_array[i * width + idx_x] = 0;
        }
    }

    // Randomly select two additional spots and apply the same logic using hiprand
    if (hiprand_uniform(&state) < 0.0005) {
        for (int i = 0; i < width; ++i) {
            maze_array[idx_y * width + i] = 0;
        }

        for (int i = 0; i < height; ++i) {
            maze_array[i * width + idx_x] = 0;
        }
    }

    maze_array[start_idx] = 2;
    maze_array[end_idx] = 3;
}

void guranteePathCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    guaranteePathKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void copyKernel(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    __shared__ int shared_data[3][3];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int size = width * height;

    int start_idx = start_y * width + start_x;
    int end_idx = end_y * width + end_x;

    // Initialize shared_data with 1's in the outside and 0's in the middle and one random spot
    if (threadIdx.x < 3 && threadIdx.y < 3)
    {
        if (threadIdx.x == 1 && threadIdx.y == 1)
        {
            shared_data[threadIdx.y][threadIdx.x] = 4; // Middle value is 0
        }
        else
        {
            shared_data[threadIdx.y][threadIdx.x] = 1; // Outside values are 1
        }
    }

    __syncthreads(); // Synchronize threads to make sure shared_data is populated

    hiprandState_t state;
    hiprand_init(seed + tid, tid, 0, &state);

    // Iterate through maze_array and copy shared_data to random indexes based on the random_value
    for (int i = tid; i < size; i += blockDim.x * gridDim.x)
    {
        // Generate a random value
        float random_value = hiprand_uniform(&state);

        // Copy shared_data to maze_array based on the random_value
        if (random_value < 0.005)
        {
            int start_index_x = i % (width - 3 + 1);
            int start_index_y = (i / width) % (height - 3 + 1);

            for (int j = 0; j < 3; ++j)
            {
                for (int k = 0; k < 3; ++k)
                {
                    maze_array[(start_index_y + j) * width + (start_index_x + k)] = shared_data[j][k];
                }
            }
        }
    }

    maze_array[start_idx] = 2;
    maze_array[end_idx] = 3;

}

void copyCuda(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    // Declare device arrays
    int* d_maze_array;

    int maze_size = width * height;

    // Allocate device memory
    hipMalloc((void**)&d_maze_array, sizeof(int) * maze_size);

    // Copy data from host to device
    hipMemcpy(d_maze_array, maze_array, sizeof(int) * maze_size, hipMemcpyHostToDevice);

    // Set up grid and block sizes
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Launch the kernel
    copyKernel << < grid, block >> > (d_maze_array, width, height, start_x, start_y, end_x, end_y, seed);

    // Copy the result back to the host
    hipMemcpy(maze_array, d_maze_array, sizeof(int) * maze_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

////////////////////////Control functions

//epsilonGreedykernel_non_async
__global__ void epsilonGreedyKernel_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {

int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < num_episodes) {
    float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
    exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
}
}

void epsilonGreedyCUDA_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    float* d_exploration_rates;

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));
    hipMemcpy(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    epsilonGreedyKernel << <grid, block >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    hipMemcpy(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_exploration_rates);
    hipDeviceSynchronize();
}

__global__ void randomArrayKernel_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

int idx = idx_y * width + idx_x;

hiprandState state;
hiprand_init(seed, idx, 0, &state);

// Set the maze value randomly
maze_array[idx] = hiprand_uniform(&state) < 0.4 ? 0 : 1;
maze_array[start_y * width + start_x] = 2;
maze_array[end_y * width + end_x] = 3;

}

void randomArrayCuda_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void randomizeZerosKernel_ctrl(int* A, int size, float percentage, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * size + idx_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    if (A[idx] == 1 && hiprand_uniform(&state) < percentage) {
        A[idx] = 0;
    }
}

void randomizeZerosCuda_ctrl(int* A, int X, int Y, float percentage, unsigned long long seed) {
    int* d_A;

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((X + block.x - 1) / block.x, (Y + block.y - 1) / block.y);

    int size = X * Y * sizeof(int);

    hipMalloc((void**)&d_A, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    randomizeZerosKernel << <grid, block >> > (d_A, X, percentage, seed);

    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipDeviceSynchronize();
}

