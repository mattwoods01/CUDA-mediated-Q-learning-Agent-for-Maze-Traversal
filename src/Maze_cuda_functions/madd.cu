#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end);
__global__ void randomArrayKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);
__global__ void randomizeZerosKernel(int* array, int size, float percentage, unsigned long long seed);
__global__ void dfs_kernel(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed, int dynamic_size);
__global__ void guranteePathKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y);

__global__ void randomArrayKernel_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);
__global__ void epsilonGreedyKernel_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end);
__global__ void dfs_kernel_ctrl(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed);

__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_episodes) {
        float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
        exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
    }
}

void epsilonGreedyCUDA(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end, hipStream_t stream = 0) {
    float* d_exploration_rates;

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice, stream);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    // Launch kernel with specified stream
    epsilonGreedyKernel << <grid, block, 0, stream >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost, stream);

    hipFree(d_exploration_rates);
}

__global__ void randomArrayKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * width + idx_x;

    __shared__ int shared_start_x, shared_start_y, shared_end_x, shared_end_y;

    // Only one thread in the block initializes shared variables
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_start_x = start_x;
        shared_start_y = start_y;
        shared_end_x = end_x;
        shared_end_y = end_y;
    }

    // Synchronize to make sure shared variables are initialized before use
    __syncthreads();

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Use shared_start_x, shared_start_y, shared_end_x, shared_end_y instead of start_x, start_y, end_x, end_y
    maze_array[idx] = hiprand_uniform(&state) < 0.4 ? 0 : 1;
    // Avoid global memory access in the loop
    if (idx_x == shared_start_x && idx_y == shared_start_y)
        maze_array[shared_start_y * width + shared_start_x] = 2;

    if (idx_x == shared_end_x && idx_y == shared_end_y)
        maze_array[shared_end_y * width + shared_end_x] = 3;
}

void randomArrayCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void randomizeZerosKernel(int* A, int size, float percentage, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * size + idx_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    if (A[idx] == 1 && hiprand_uniform(&state) < percentage) {
        A[idx] = 0;
    }
}

void randomizeZerosCuda(int* A, int X, int Y, float percentage, unsigned long long seed) {
    int* d_A;

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((X + block.x - 1) / block.x, (Y + block.y - 1) / block.y);

    int size = X * Y * sizeof(int);

    hipMalloc((void**)&d_A, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    randomizeZerosKernel << <grid, block >> > (d_A, X, percentage, seed);

    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipDeviceSynchronize();
}

__device__ void swap(int& a, int& b) {
    int temp = a;
    a = b;
    b = temp;
}

__global__ void dfs_kernel(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed, int dynamic_size) {
    int current_x = blockIdx.x * blockDim.x + threadIdx.x;
    int current_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = current_y * width + current_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Shared memory to store information between threads within a block
    __shared__ int shared_end_reached;
    extern __shared__ int shared_maze_array[];

    // Initialize shared_end_reached only once
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_end_reached = 0;
    }

    // Synchronize to ensure proper initialization
    __syncthreads();

    // Check if the current cell is the end cell
    if ((current_x == end_x && current_y == end_y) || (current_x == start_x && current_y == start_y)) {
        // Mark the end point as reached
        shared_end_reached = 1;
    }

    // Synchronize to ensure all threads have marked the end point
    __syncthreads();

    // Check if either endpoint has been reached
    if (shared_end_reached) {
        // You can handle the end condition here or use it as a flag for further operations
        return;
    }

    // Define the possible moves (right, left, down, up)
    int moves[4][2] = { {0, 1}, {0, -1}, {1, 0}, {-1, 0} };

    // Fisher-Yates shuffle to traverse randomly
    for (int i = 3; i > 0; --i) {
        int j = hiprand_uniform(&state) * (i + 1);

        // Swap moves[i] with moves[j]
        swap(moves[i][0], moves[j][0]);
        swap(moves[i][1], moves[j][1]);
    }

    // Check each possible move
    for (int i = 0; i < 4; ++i) {
        int new_x = current_x + moves[i][0];
        int new_y = current_y + moves[i][1];

        // Check if the new position is within bounds
        if (new_x >= 0 && new_x < width && new_y >= 0 && new_y < height) {
            int new_idx = new_y * width + new_x;

            // Check if the new cell is open and not visited
            if (maze_array[new_idx] == 1) {
                // If another thread marked the end point, stop the traversal
                if (shared_maze_array[end_y * width + end_x] == 4) {
                    return;
                }

                if (hiprand_uniform(&state) < 0.4) {
                    // Use shared_maze_array to communicate information between threads
                    atomicExch(&shared_maze_array[new_idx], 0);
                }

                // Recursively call DFS on the new cell
                dfs_kernel << <1, 1, dynamic_size * sizeof(int) >> > (maze_array, width, height, start_x, start_y, end_x, end_y, seed, dynamic_size);

                // If the end has been reached in the recursive call, update shared_maze_array
                if (maze_array[end_y * width + end_x] == 4) {
                    atomicExch(&shared_maze_array[end_y * width + end_x], 4);
                    return;
                }
            }
        }
    }

}

void dfsCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;
    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;

    int dynamic_size = width * height;

    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    dfs_kernel << <block, grid, dynamic_size * sizeof(int) >> > (d_maze_array, width, height, start_x, start_y, end_x, end_y, seed, dynamic_size);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void guranteePathKernel(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int start_idx = start_y * width + start_x;
    int end_idx = end_y * width + end_x;

    // Set cells in the same row as start to 0 for only half of the width
    if (idx_y == start_y) {
        for (int i = 0; i < width / 2; ++i) {
            maze_array[idx_y * width + i] = 0;
        }
    }

    // Set cells in the same row as end to 0 for only half of the width
    if (idx_y == end_y) {
        for (int i = width / 2; i < width; ++i) {
            maze_array[idx_y * width + i] = 0;
        }
    }

    // Set cells in the same column as start to 0 for only half of the height
    if (idx_x == start_x) {
        for (int i = 0; i < height / 2; ++i) {
            maze_array[i * width + idx_x] = 0;
        }
    }

    // Set cells in the same column as end to 0 for only half of the height
    if (idx_x == end_x) {
        for (int i = height / 2; i < height; ++i) {
            maze_array[i * width + idx_x] = 0;
        }
    }

    maze_array[start_idx] = 3;
    maze_array[end_idx] = 4;
}

void guranteePathCuda(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    guranteePathKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}


///////////////////////////Control functions

//epsilonGreedykernel_non_async
__global__ void epsilonGreedyKernel_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {

int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < num_episodes) {
    float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
    exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
}
}

void epsilonGreedyCUDA_ctrl(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    float* d_exploration_rates;

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));
    hipMemcpy(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    epsilonGreedyKernel << <grid, block >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    hipMemcpy(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_exploration_rates);
    hipDeviceSynchronize();
}

//randomArrayKernel_non_shared
__global__ void randomArrayKernel_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

int idx = idx_y * width + idx_x;

hiprandState state;
hiprand_init(seed, idx, 0, &state);

// Set the maze value randomly
maze_array[idx] = hiprand_uniform(&state) < 0.3 ? 0 : 1;
maze_array[start_y * width + start_x] = 2;
maze_array[end_y * width + end_x] = 3;

}

void randomArrayCuda_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

//dfs_kernel_non_shared
__global__ void dfs_kernel_ctrl(int* maze_array, int width, int height, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int current_x = blockIdx.x * blockDim.x + threadIdx.x;
    int current_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = current_y * width + current_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Check if the current cell is the end cell
    if (current_x == end_x && current_y == end_y) {
        // You've reached the end, you can handle it as needed
        return;
    }

    // Define the possible moves (right, left, down, up)
    int moves[4][2] = { {0, height / 2}, {0, -height / 2}, {width / 2, 0}, {-width / 2, 0} };

    // Fisher-Yates shuffle to traverse randomly
    for (int i = 3; i > 0; --i) {
        int j = hiprand_uniform(&state) * (i + 1);

        // Swap moves[i] with moves[j]
        swap(moves[i][0], moves[j][0]);
        swap(moves[i][1], moves[j][1]);
    }

    // Check each possible move
    for (int i = 0; i < 4; ++i) {
        int new_x = current_x + moves[i][0];
        int new_y = current_y + moves[i][1];

        // Check if the new position is within bounds
        if (new_x >= 0 && new_x < width && new_y >= 0 && new_y < height) {
            int new_idx = new_y * width + new_x;


            // Check if the new cell is open and not visited
            if (maze_array[new_idx] == 1) {
                if (hiprand_uniform(&state) < 0.2) {
                    maze_array[new_idx] = 0;
                }
                // Recursively call DFS on the new cell
                dfs_kernel_ctrl << < 1, 1 >> > (maze_array, width, height, start_x, start_y, end_x, end_y, seed);

                // If the end has been reached in the recursive call, exit the loop
                if (maze_array[end_y * width + end_x] == 4) {
                    return;
                }
            }
        }
    }

    // Additional condition to prevent changing the starting cell

}

void dfsCuda_ctrl(int* maze_array, int height, int width, int start_x, int start_y, int end_x, int end_y, unsigned long long seed) {
    int* d_maze_array;
    hipMalloc((void**)&d_maze_array, height * width * sizeof(int));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(int), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    dfs_kernel_ctrl << <grid, block >> > (d_maze_array, width, height, start_x, start_y, end_x, end_y, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}