#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end);
__global__ void randomArrayKernel(int* maze_array, int height, int width, unsigned long long seed);
__global__ void update_q_table_kernel(float* q_value, float next_q_value, int state_x, int state_y, int action, int next_state_x, int next_state_y, float reward, float learning_rate, float discount_factor);
__global__ void randomizeZerosKernel(int* array, int size, float percentage, unsigned long long seed);

__global__ void epsilonGreedyKernel(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_episodes) {
        float frac = static_cast<float>(tid) / static_cast<float>(num_episodes);
        exploration_rates[tid] = exploration_start * exp(frac * log(exploration_end / exploration_start));
    }
}

void epsilonGreedyCUDA(float* exploration_rates, int num_episodes, float exploration_start, float exploration_end) {
    float* d_exploration_rates;

    hipMalloc((void**)&d_exploration_rates, num_episodes * sizeof(float));
    hipMemcpy(d_exploration_rates, exploration_rates, num_episodes * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int blocks_per_grid = (num_episodes + dimx - 1) / dimx;
    dim3 block(dimx, 1);
    dim3 grid(blocks_per_grid, 1);

    epsilonGreedyKernel << <grid, block >> > (d_exploration_rates, num_episodes, exploration_start, exploration_end);

    hipMemcpy(exploration_rates, d_exploration_rates, num_episodes * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_exploration_rates);
    hipDeviceSynchronize();
}

__device__ void dfs(int* maze_array, int height, int width, int x, int y, hiprandState* state) {
    // Mark the current cell as visited
    maze_array[y * width + x] = 0;

    // Define possible directions (up, down, left, right)
    int directions[4][2] = { {0, -1}, {0, 1}, {-1, 0}, {1, 0} };

    // Shuffle the directions randomly
    for (int i = 0; i < 4; ++i) {
        int rand_idx = static_cast<int>(hiprand_uniform(state) * 2);
        int temp_x = x + directions[rand_idx][0];
        int temp_y = y + directions[rand_idx][1];

        // Check if the new cell is within bounds
        if (temp_x >= 0 && temp_x < width && temp_y >= 0 && temp_y < height) {
            // Check if the neighboring cell is visited
            if (maze_array[temp_y * width + temp_x] == 1) {
                // Recursively call dfs for the adjacent cell
                maze_array[temp_y * width + temp_x] = 0;
                maze_array[(y + temp_y) / 2 * width + (x + temp_x) / 2] = 0;
                dfs(maze_array, height, width, temp_x, temp_y, state);
                
            }
        }

    }
}

__global__ void randomArrayKernel(int* maze_array, int height, int width, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * width + idx_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Set the maze value randomly
    maze_array[idx] = hiprand_uniform(&state) < 0.55 ? 0 : 1;

    // Ensure that the values surrounding the first and last indices are 0
    if ((idx_x == 0 || idx_x == width - 1) && (idx_y == 0 || idx_y == height - 1)) {
        // Set neighboring values to 0
        maze_array[idx] = 0;
        maze_array[idx + 1] = 0;  // Right neighbor
        maze_array[idx - 1] = 0;  // Left neighbor
        maze_array[idx + width] = 0;  // Bottom neighbor
        maze_array[idx - width] = 0;  // Top neighbor
    }

    // Ensure connectivity by applying DFS from the top-left corner
    if (idx_x == 0 && idx_y == 0) {
        dfs(maze_array, height, width, idx_x, idx_y, &state);
    }
}

void randomArrayCuda(int* maze_array, int height, int width, unsigned long long seed) {
    int* d_maze_array;

    hipMalloc((void**)&d_maze_array, height * width * sizeof(float));
    hipMemcpy(d_maze_array, maze_array, height * width * sizeof(float), hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    randomArrayKernel << <grid, block >> > (d_maze_array, height, width, seed);

    hipMemcpy(maze_array, d_maze_array, height * width * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_maze_array);
    hipDeviceSynchronize();
}

__global__ void update_q_table_kernel(float* q_value, float next_q_value, int state_x, int state_y, int action, int next_state_x, int next_state_y, float reward, float learning_rate, float discount_factor) {
    // Calculate the index directly, assuming q_value is a single value

    *q_value = *q_value + learning_rate * (reward + discount_factor * next_q_value - *q_value);
}

float* d_q_value = nullptr;

void update_q_table_cuda(float* q_value, float next_q_value, int state_x, int state_y, int action, int next_state_x, int next_state_y, float reward, float learning_rate, float discount_factor) {
    // Check if device memory is allocated, if not, allocate it
    if (!d_q_value) {
        hipMalloc((void**)&d_q_value, sizeof(float));
    }

    // Copy the input q_value to the device
    hipMemcpy(d_q_value, q_value, sizeof(float), hipMemcpyHostToDevice);

    // Use a 1D grid and block for simplicity, as you are dealing with a single value
    dim3 block(1);
    dim3 grid(1);

    // Update q_value on the device
    update_q_table_kernel << <grid, block >> > (d_q_value, next_q_value, state_x, state_y, action, next_state_x, next_state_y, reward, learning_rate, discount_factor);

    // Synchronize the device
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(q_value, d_q_value, sizeof(float), hipMemcpyDeviceToHost);
}

void cleanup_cuda() {
    if (d_q_value != nullptr) {
        hipFree(d_q_value);
        d_q_value = nullptr;
    }
}

__global__ void randomizeZerosKernel(int* A, int size, float percentage, unsigned long long seed) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = idx_y * size + idx_x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    if (A[idx] == 1 && hiprand_uniform(&state) < percentage) {
        A[idx] = 2;
    }
}

void randomizeZerosCuda(int* A, int X, int Y, float percentage, unsigned long long seed) {
    int* d_A;

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((X + block.x - 1) / block.x, (Y + block.y - 1) / block.y);

    int size = X * Y * sizeof(int);  

    hipMalloc((void**)&d_A, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    randomizeZerosKernel << <grid, block >> > (d_A, X, percentage, seed);  

    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipDeviceSynchronize();
}

